#include "hip/hip_runtime.h"
/* Matrix multiplication: P = M * N.
bal__
void MatrixMulKernel(float* MD, float Nd, float* Pd, int Width){
//calculate row index of the Pd element and M
	int Row = blockIdx.y*blockDim.y+threadIdx.y;
//calculate column index of Pd element and 	int Col = blockIdx.x*blockDim.x+threadIdx.x;
	if((Row < Width) && (Col < Width)){
		float Pvalue = 0.0;
		for(int k = 0; k < Width; ++k)
			Pvalue += Md[Row*Width+K] *Nd[k*Width+Col];
		PD[Row*Width+Col] = Pvalue;
		}
}
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"

// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
int Width = M.width; 
int Row = blockIdx.y*blockDim.y+threadIdx.y;
//calculate column index of Pd element and     
int Col = blockIdx.x*blockDim.x+threadIdx.x;
if((Row < Width) && (Col < Width)){
        float Pvalue = 0.0;
        for(int k = 0; k < Width; k++)
                Pvalue += M.elements[Row*Width+k]*N.elements[k*Width+Col];
                P.elements[Row*Width+Col] = Pvalue;
                }

}



#endif // #ifndef _MATRIXMUL_KERNEL_H_
